#define NEPOCHS 1000
#define DIMENSIONX 1000
#define DIMENSIONY 1000


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <fstream>

__global__
void Evolve(bool *u, int n, int dx, int dy)
{
    int entry_index = blockIdx.x*blockDim.x + threadIdx.x;
    if (entry_index>=dx*dy) return;
    
    int i = entry_index % dy;
    int j = entry_index / dy;

    //get number of neighbors
    size_t NActiveNeighbors = 0;
    int nmin_x = max(0, i - 1);
    int nmax_x = min(i + 1, dx - 1);
    int nmin_y = max(0, j - 1);
    int nmax_y = min(j + 1, dy - 1);
    for (int ii = nmin_x; ii <= nmax_x; ii++)
        for (int jj = nmin_y; jj <= nmax_y; jj++)
        {
            if ((i == ii) && (j == jj))
                continue;
            if (u[n*dx*dy+ii*dx+jj]) NActiveNeighbors += 1;
        }

    bool active_pre, active_post;
    int n_next = n + 1;
        
    active_pre = u[n*dx*dy+i*dx+j];
    active_post = false;
    if (active_pre && (NActiveNeighbors == 2))
        active_post = true;
    else if (active_pre && (NActiveNeighbors == 3))
        active_post = true;
    else if ((!active_pre) && (NActiveNeighbors == 3))
        active_post = true;
    u[n_next*dx*dy+i*dx+j] = active_post;

    return;
}

bool *allocate_universe(int n, int dx, int dy)
{
    int Nelements = n*dx*dy;
    bool *universe = new bool[Nelements];
    for (int i = 0; i < Nelements; i++)
    {
      universe[i] = false;
    }
    return universe;
}

void set_initial_conditions(bool *u, int dx, int dy, float p)
{
    for (int i = 0; i < dx; i++)
        for (int j = 0; j < dy; j++)
        {
            if (((rand() % 1000) < p * 1000))
            {
                u[i*dx+j] = true;
            }
        }
    rand();
}

int main(void)
{
  long int NENTRIES = NEPOCHS*DIMENSIONX*DIMENSIONY;
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  std::cout << "Defining the universe and creating the initial conditions" << std::endl;
  
  bool *universe = allocate_universe(NEPOCHS, DIMENSIONX, DIMENSIONY);
  set_initial_conditions(universe, DIMENSIONX, DIMENSIONY, 0.25);
  
  bool *cuda_universe;
  hipMalloc((void**)&cuda_universe, NENTRIES*sizeof(bool));
  hipMemcpy(cuda_universe, universe, NENTRIES*sizeof(bool), hipMemcpyHostToDevice);
  hipEventRecord(start);
  for (int n=0; n<NEPOCHS-1; n++) {
    if (n % 10 == 0)
    std::cout << n << "/" << NEPOCHS << std::endl;
    Evolve<<<1+DIMENSIONX*DIMENSIONY/256, 256>>>(cuda_universe, n, DIMENSIONX, DIMENSIONY);
  }
  hipEventRecord(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Total execution time (ms): %f\n ", milliseconds);
  hipMemcpy(universe, cuda_universe, NENTRIES*sizeof(bool), hipMemcpyDeviceToHost);
  
  //write everything to file  
  std::ofstream outfile;
  outfile.open ("evolution.txt");
  outfile << NEPOCHS << " , " << DIMENSIONX << " , " << DIMENSIONY << "\n";
  int nepoch, nx, ny;
  for (int i=0; i<NENTRIES; i++) {
    if (universe[i]) {
      nepoch = i/(DIMENSIONX*DIMENSIONY);
      nx = (i % (DIMENSIONX*DIMENSIONY)) / DIMENSIONX;
      ny = i % DIMENSIONX;
      outfile << nepoch << " , " << nx << " , " << ny << "\n";
    }
  }
  outfile.close();
  
  //clean up
  hipFree(cuda_universe);
  free(universe);
}